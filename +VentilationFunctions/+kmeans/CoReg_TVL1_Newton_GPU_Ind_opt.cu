#include "hip/hip_runtime.h"
/***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <mex.h>
#include <math.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "CoReg_TVL1_Newton_Ind_kernels.cu"

#define BLOCKSIZE 512
#define MAX_GRID_SIZE 65535
#define NUMTHREADS 512

dim3 GetGrid(int size){
    size = (size-1) / NUMTHREADS + 1;
    dim3 grid( size, 1, 1 );
    if( grid.x > MAX_GRID_SIZE ) grid.x = grid.y = (int) sqrt( (double)(size-1) ) + 1;
    else if( grid.y > MAX_GRID_SIZE ) grid.x = grid.y = grid.z = (int) pow( (double)(size-1), (double)1.0/3.0 ) + 1;
    return grid;
}

extern void mexFunction(int iNbOut, mxArray *pmxOut[],
        int iNbIn, const mxArray *pmxIn[]){
    
    /* iNbOut: number of outputs */
    /* pmxOut: array of pointers to output arguments */
    
    /* iNbIn: number of inputs
    /* pmxIn: array of pointers to input arguments */
    
    /*  host arrays and variables */
    float   *h_u1x, *h_u1y, *h_u1z, *h_cvg, *h_U1x, *h_U1y, *h_U1z;
    float   *h_u2x, *h_u2y, *h_u2z, *h_U2x, *h_U2y, *h_U2z;
    float   *h_VecParameters,*h_G1x, *h_G1y, *h_G1z, *h_G1f, *h_G1t;
    float   *h_G2x, *h_G2y, *h_G2z, *h_G2f, *h_G2t;
    float   *h_b1x1, *h_b1x2, *h_b1x3, *h_b1y1, *h_b1y2, *h_b1y3, *h_b1z1, *h_b1z2, *h_b1z3;
    float   *h_b2x1, *h_b2x2, *h_b2x3, *h_b2y1, *h_b2y2, *h_b2y3, *h_b2z1, *h_b2z2, *h_b2z3;
    float   *h_q1, *h_q2, *h_gkx, *h_gky, *h_gkz, *tt, *h_dv1x, *h_dv1y, *h_dv1z;
    float   *h_dv2x, *h_dv2y, *h_dv2z;
    float   fError, cc, steps, fPenalty1, fPenalty2, fPenalty3, fps;
    float   weight1, weight2;
    /*
    int     *punum, iNy, iNx, iNz, iNdim, iDim[3], iNI;
    int     maxIter, SZF, iDev;
    */
    int     *punum, iNy, iNx, iNz, iNdim, iDim[3], maxIter;
    int     flag = 1;
    
    
    hipSetDevice(1);
    
    /* Timing */
    hipEvent_t start, stop;
    float time;
    
    /*  device arrays */
    float   *d_b1x1, *d_b1y1, *d_b1z1, *d_b1x2, *d_b1y2, *d_b1z2, *d_b1x3, *d_b1y3, *d_b1z3, *d_dv1x, *d_dv1y, *d_dv1z;
    float   *d_b2x1, *d_b2y1, *d_b2z1, *d_b2x2, *d_b2y2, *d_b2z2, *d_b2x3, *d_b2y3, *d_b2z3, *d_dv2x, *d_dv2y, *d_dv2z;

    float   *d_q1, *d_q2;
    float   *d_u1x, *d_u1y, *d_u1z, *d_U1x, *d_U1y, *d_U1z, *d_G1x, *d_G1y, *d_G1z, *d_G1f, *d_G1t;
    float   *d_u2x, *d_u2y, *d_u2z, *d_U2x, *d_U2y, *d_U2z, *d_G2x, *d_G2y, *d_G2z, *d_G2f, *d_G2t;
    float   *d_gkx, *d_gky, *d_gkz;

    float   *h_FPS, *d_FPS;
    float   *h_qq1, *h_qq2,*h_qq3, *d_qq1, *d_qq2, *d_qq3;
    
    /* CUDA event-based timer start */
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );
    
    /* input interface with matlab arrays */
    h_VecParameters = (float *)mxGetData(pmxIn[0]); /* Vector of parameters */
    h_U1x = (float *)mxGetData(pmxIn[1]);
    h_U1y = (float *)mxGetData(pmxIn[2]);
    h_U1z = (float *)mxGetData(pmxIn[3]);
    h_G1x = (float *)mxGetData(pmxIn[4]);
    h_G1y = (float *)mxGetData(pmxIn[5]);
    h_G1z = (float *)mxGetData(pmxIn[6]);
    h_G1t = (float *)mxGetData(pmxIn[7]);
    h_G1f = (float *)mxGetData(pmxIn[8]);

    h_U2x = (float *)mxGetData(pmxIn[9]);
    h_U2y = (float *)mxGetData(pmxIn[10]);
    h_U2z = (float *)mxGetData(pmxIn[11]);
    h_G2x = (float *)mxGetData(pmxIn[12]);
    h_G2y = (float *)mxGetData(pmxIn[13]);
    h_G2z = (float *)mxGetData(pmxIn[14]);
    h_G2t = (float *)mxGetData(pmxIn[15]);
    h_G2f = (float *)mxGetData(pmxIn[16]);
    
    
    /* dimensions */
    iNy = (int) h_VecParameters[0];
    iNx = (int) h_VecParameters[1];
    iNz = (int) h_VecParameters[2];
    
    unsigned int imageSize = iNx*iNy*iNz;
    
    /* parameters */
    maxIter = (int) h_VecParameters[3]; /* total number of iterations */
    fError = (float) h_VecParameters[4]; /* error criterion */
    cc = (float) h_VecParameters[5]; /* cc for ALM */
    steps = (float) h_VecParameters[6]; /* steps for each iteration */
    fPenalty1 = (float) h_VecParameters[7];
    fPenalty2 = (float) h_VecParameters[8];
    fPenalty3 = (float) h_VecParameters[9];
    weight1 = (float) h_VecParameters[10];
    weight2 = (float) h_VecParameters[11];
    
    /* output interface with matlab */
    /* u1x */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    pmxOut[0] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_u1x = (float*)mxGetData(pmxOut[0]);
    
    /* u1y */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    
    pmxOut[1] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_u1y = (float*)mxGetData(pmxOut[1]);
    
    /* u1z */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    pmxOut[2] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_u1z = (float*)mxGetData(pmxOut[2]);

    /* u2x */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    pmxOut[3] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_u2x = (float*)mxGetData(pmxOut[3]);
    
    /* u2y */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    
    pmxOut[4] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_u2y = (float*)mxGetData(pmxOut[4]);
    
    /* u2z */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    pmxOut[5] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_u2z = (float*)mxGetData(pmxOut[5]);
    
    /* convergence rate */
    iNdim = 2;
    iDim[0] = 1;
    iDim[1] = maxIter;
    pmxOut[6] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_cvg = (float*)mxGetData(pmxOut[6]);
    
    /* number of iterations */
    iNdim = 2;
    iDim[0] = 1;
    iDim[1] = 1;
    pmxOut[7] = mxCreateNumericArray(iNdim,(const int*)iDim,mxUINT16_CLASS,mxREAL);
    punum = (int*)mxGetData(pmxOut[7]);
    
    /* computation time */
    iNdim = 2;
    iDim[0] = 1;
    iDim[1] = 1;
    pmxOut[8] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    tt = (float*)mxGetData(pmxOut[8]);
    
    /* allocate host memory */
    /* bx1, bx2, bx3 */
    h_b1x1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b1x2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b1x3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_b1x1 || !h_b1x2 || !h_b1x3) mexPrintf("calloc: Memory allocation failure\n");
    
    /* by1, by2, by3 */
    h_b1y1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b1y2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b1y3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_b1y1 || !h_b1y2 || !h_b1y3) mexPrintf("calloc: Memory allocation failure\n");
    
    /* bz1, bz2, bz3 */
    h_b1z1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b1z2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b1z3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_b1z1 || !h_b1z2 || !h_b1z3) mexPrintf("calloc: Memory allocation failure\n");

    /* bx1, bx2, bx3 */
    h_b2x1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b2x2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b2x3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_b2x1 || !h_b2x2 || !h_b2x3) mexPrintf("calloc: Memory allocation failure\n");
    
    /* by1, by2, by3 */
    h_b2y1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b2y2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b2y3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_b2y1 || !h_b2y2 || !h_b2y3) mexPrintf("calloc: Memory allocation failure\n");
    
    /* bz1, bz2, bz3 */
    h_b2z1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b2z2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_b2z3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_b2z1 || !h_b2z2 || !h_b2z3) mexPrintf("calloc: Memory allocation failure\n");
    
    
    /* q1 */
    h_q1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_q1) mexPrintf("calloc: Memory allocation failure\n");
    
    /* q2 */
    h_q2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_q2) mexPrintf("calloc: Memory allocation failure\n");
    
    /* gk */
    h_gkx = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_gky = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_gkz = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_gkx || !h_gky || !h_gkz) mexPrintf("calloc: Memory allocation failure\n");
    
    /* div */
    h_dv1x = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_dv1y = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_dv1z = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_dv1x || !h_dv1y || !h_dv1z ) mexPrintf("calloc: Memory allocation failure\n");

    /* div */
    h_dv2x = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_dv2y = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_dv2z = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_dv2x || !h_dv2y || !h_dv2z ) mexPrintf("calloc: Memory allocation failure\n");
    

    /*I am here now~~. continue tomorrow.*/
    /* h_FPS */
    h_FPS = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_FPS) mexPrintf("calloc: Memory allocation failure\n");

    /* h_qq1 */
    h_qq1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_qq1) mexPrintf("calloc: Memory allocation failure\n");
    
    /* h_qq2 */
    h_qq2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_qq2) mexPrintf("calloc: Memory allocation failure\n");

    /* h_qq3 */
    h_qq3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_qq3) mexPrintf("calloc: Memory allocation failure\n");
    
    
    
    /* device memory allocation */
    hipMalloc( (void**) &d_b1x1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b1x2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b1x3, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b1y1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b1y2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b1y3, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b1z1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b1z2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b1z3, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_gkx, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_gky, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_gkz, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_dv1x, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_dv1y, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_dv1z, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_q1,  sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_u1x, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_u1y, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_u1z, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_U1x, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_U1y, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_U1z, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G1x, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G1y, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G1z, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G1t, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G1f, sizeof(float)*(unsigned)imageSize);

    hipMalloc( (void**) &d_b2x1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b2x2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b2x3, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b2y1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b2y2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b2y3, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b2z1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b2z2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_b2z3, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_dv2x, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_dv2y, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_dv2z, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_q2,  sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_u2x, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_u2y, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_u2z, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_U2x, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_U2y, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_U2z, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G2x, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G2y, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G2z, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G2t, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_G2f, sizeof(float)*(unsigned)imageSize);
    
    hipMalloc( (void**) &d_FPS, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_qq1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_qq2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_qq3, sizeof(float)*(unsigned)imageSize);
    
    /* copy arrays from host to device */
    hipMemcpy( d_b1x1, h_b1x1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b1x2, h_b1x2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b1x3, h_b1x3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b1y1, h_b1y1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b1y2, h_b1y2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b1y3, h_b1y3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b1z1, h_b1z1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b1z2, h_b1z2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b1z3, h_b1z3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_gkx, h_gkx, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_gky, h_gky, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_gkz, h_gkz, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_dv1x, h_dv1x, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_dv1y, h_dv1y, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_dv1z, h_dv1z, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_q1,  h_q1,  sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_u1x, h_u1x, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_u1y, h_u1y, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_u1z ,h_u1z, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_U1x, h_U1x, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_U1y, h_U1y, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_U1z, h_U1z, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G1x, h_G1x, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G1y, h_G1y, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G1z, h_G1z, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G1t, h_G1t, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G1f, h_G1f, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);

    hipMemcpy( d_b2x1, h_b2x1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b2x2, h_b2x2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b2x3, h_b2x3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b2y1, h_b2y1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b2y2, h_b2y2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b2y3, h_b2y3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b2z1, h_b2z1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b2z2, h_b2z2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_b2z3, h_b2z3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_dv2x, h_dv2x, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_dv2y, h_dv2y, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_dv2z, h_dv2z, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_q2,  h_q2,  sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_u2x, h_u2x, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_u2y, h_u2y, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_u2z ,h_u2z, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_U2x, h_U2x, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_U2y, h_U2y, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_U2z, h_U2z, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G2x, h_G2x, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G2y, h_G2y, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G2z, h_G2z, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G2t, h_G2t, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_G2f, h_G2f, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    
    hipMemcpy( d_qq1, h_qq1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_qq2, h_qq2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_qq3, h_qq3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    
    
    /* run optimization */
    
    /* iNI = 0; */
    dim3 threads(BLOCKSIZE,1,1);
    dim3 grid = GetGrid(imageSize);
    
    for( int i = 0; i < maxIter; i++)
    {
        
        /* update p1 */
        krnl_1<<<grid, threads>>>(d_dv1x, d_dv1y, d_dv1z, 
                d_u1x, d_u1y, d_u1z,
                d_gkx, d_gky, d_gkz, 
                d_G1x, d_G1y, d_G1z,
                d_G1t, d_G1f, d_q1, 
                d_U1x, d_U1y, d_U1z,
                cc, iNx, iNy, iNz, d_qq1, d_qq2, d_qq3, flag, weight1);

        /* update p1x p1y, p1z */
        krnl_23z<<<grid, threads>>>(d_b1x1, d_b1y1, d_b1z1, 
                d_b1x2, d_b1y2, d_b1z2,
                d_b1x3, d_b1y3, d_b1z3,
                d_gkx, d_gky, d_gkz,
                steps, iNx, iNy, iNz);
        /*
//         krnl_2<<<grid, threads>>>(d_bx1, d_by1, d_bz1, d_gkx, d_gky, d_gkz,
//                 steps, iNx, iNy, iNz);
// 
//         krnl_3<<<grid, threads>>>(d_bx2, d_by2, d_bz2, d_gkx, d_gky, d_gkz,
//                 steps, iNx, iNy, iNz);
//         
//         krnl_z<<<grid, threads>>>(d_bx3, d_by3, d_bz3, d_gkx, d_gky, d_gkz,
//                 steps, iNx, iNy, iNz);
        */
        
        /* projection step */
        krnl_4<<<grid, threads>>>(d_b1x1, d_b1x2, d_b1x3, 
                d_b1y1, d_b1y2, d_b1y3,
                d_b1z1, d_b1z2, d_b1z3, 
                d_gkx, d_gky, d_gkz,
                fPenalty1*weight1, iNx, iNy, iNz);
        
        krnl_56zp<<<grid, threads>>>(d_b1x1, d_b1y1, d_b1z1,
                d_b1x2, d_b1y2, d_b1z2,
                d_b1x3, d_b1y3, d_b1z3,
                d_gkx, d_gky, d_gkz,
                iNx, iNy, iNz);
        /*
//         krnl_5<<<grid, threads>>>(d_bx1, d_by1, d_bz1, d_gkx, d_gky, d_gkz,
//                 iNx, iNy, iNz);
//         
//         krnl_6<<<grid, threads>>>(d_bx2, d_by2, d_bz2, d_gkx, d_gky, d_gkz,
//                 iNx, iNy, iNz);
//         
//         krnl_zp<<<grid, threads>>>(d_bx3, d_by3, d_bz3, d_gkx, d_gky, d_gkz,
//                 iNx, iNy, iNz);
        */
        krnl_7<<<grid, threads>>>(d_b1x1, d_b1x2, d_b1x3, 
                d_b1y1, d_b1y2, d_b1y3,
                d_b1z1, d_b1z2, d_b1z3, 
                d_dv1x, d_dv1y, d_dv1z, 
                iNx, iNy, iNz);
        
        /* update p2 */
        krnl_1<<<grid, threads>>>(d_dv2x, d_dv2y, d_dv2z, 
                d_u2x, d_u2y, d_u2z,
                d_gkx, d_gky, d_gkz, 
                d_G2x, d_G2y, d_G2z,
                d_G2t, d_G2f, d_q2, 
                d_U2x, d_U2y, d_U2z,
                cc, iNx, iNy, iNz, d_qq1, d_qq2, d_qq3, -flag, weight2);

        /* update p2x, p2y, p2z */
        krnl_23z<<<grid, threads>>>(d_b2x1, d_b2y1, d_b2z1, 
                d_b2x2, d_b2y2, d_b2z2,
                d_b2x3, d_b2y3, d_b2z3,
                d_gkx, d_gky, d_gkz,
                steps, iNx, iNy, iNz);
        /*
//         krnl_2<<<grid, threads>>>(d_bx1, d_by1, d_bz1, d_gkx, d_gky, d_gkz,
//                 steps, iNx, iNy, iNz);
// 
//         krnl_3<<<grid, threads>>>(d_bx2, d_by2, d_bz2, d_gkx, d_gky, d_gkz,
//                 steps, iNx, iNy, iNz);
//         
//         krnl_z<<<grid, threads>>>(d_bx3, d_by3, d_bz3, d_gkx, d_gky, d_gkz,
//                 steps, iNx, iNy, iNz);
        */
        
        /* projection step */
        krnl_4<<<grid, threads>>>(d_b2x1, d_b2x2, d_b2x3, 
                d_b2y1, d_b2y2, d_b2y3,
                d_b2z1, d_b2z2, d_b2z3, 
                d_gkx, d_gky, d_gkz,
                fPenalty2*weight2, iNx, iNy, iNz);
        
        krnl_56zp<<<grid, threads>>>(d_b2x1, d_b2y1, d_b2z1,
                d_b2x2, d_b2y2, d_b2z2,
                d_b2x3, d_b2y3, d_b2z3,
                d_gkx, d_gky, d_gkz,
                iNx, iNy, iNz);
        /*
//         krnl_5<<<grid, threads>>>(d_bx1, d_by1, d_bz1, d_gkx, d_gky, d_gkz,
//                 iNx, iNy, iNz);
//         
//         krnl_6<<<grid, threads>>>(d_bx2, d_by2, d_bz2, d_gkx, d_gky, d_gkz,
//                 iNx, iNy, iNz);
//         
//         krnl_zp<<<grid, threads>>>(d_bx3, d_by3, d_bz3, d_gkx, d_gky, d_gkz,
//                 iNx, iNy, iNz);
        */
        krnl_7<<<grid, threads>>>(d_b2x1, d_b2x2, d_b2x3, 
                d_b2y1, d_b2y2, d_b2y3,
                d_b2z1, d_b2z2, d_b2z3, 
                d_dv2x, d_dv2y, d_dv2z, 
                iNx, iNy, iNz);
        
       /* compute qq */
       krnl_8<<<grid, threads>>>(d_U1x, d_U1y, d_U1z, d_U2x, d_U2y, d_U2z,
                                 d_q1, d_q2,
                                 d_G1x, d_G1y, d_G1z, d_G2x, d_G2y, d_G2z,
                                 d_dv1x, d_dv1y, d_dv1z, d_dv2x, d_dv2y, d_dv2z, 
                                 d_u1x, d_u1y, d_u1z, d_u2x, d_u2y, d_u2z,
                                 cc, fPenalty3, iNx, iNy, iNz, d_qq1, d_qq2, d_qq3);
       /*compute u1,2x, u1,2y, u1,2z*/
       krnl_9<<<grid, threads>>>(d_dv1x, d_dv1y, d_dv1z, d_dv2x, d_dv2y, d_dv2z, 
                                 d_G1x, d_G1y, d_G1z, d_G2x, d_G2y, d_G2z,
                                 d_q1, d_q2, d_u1x, d_u1y, d_u1z, d_u2x, d_u2y, d_u2z,
                                 d_FPS, cc, iNx, iNy, iNz, d_qq1, d_qq2, d_qq3);
       
        
        /* compute convergence */
        hipMemcpy( h_FPS, d_FPS, sizeof(float)*unsigned(imageSize), hipMemcpyDeviceToHost);
        
        fps = 0;
        for (int j=0; j< imageSize; j++){
            fps += abs(h_FPS[j]);
        }
        
        h_cvg[i] = fps / (float)imageSize /6;
        
        if (h_cvg[i] <= fError){
            break; 
        }
        
        /*mexPrintf("cvg: %f\n",h_cvg[i]); */
        
        punum[0] = i+1;
        
    }
    
    /* copy arrays from device to host */
    hipMemcpy( h_u1x, d_u1x, sizeof(float)*(unsigned)(imageSize), hipMemcpyDeviceToHost);
    hipMemcpy( h_u1y, d_u1y, sizeof(float)*(unsigned)(imageSize), hipMemcpyDeviceToHost);
    hipMemcpy( h_u1z, d_u1z, sizeof(float)*(unsigned)(imageSize), hipMemcpyDeviceToHost);
    hipMemcpy( h_u2x, d_u2x, sizeof(float)*(unsigned)(imageSize), hipMemcpyDeviceToHost);
    hipMemcpy( h_u2y, d_u2y, sizeof(float)*(unsigned)(imageSize), hipMemcpyDeviceToHost);
    hipMemcpy( h_u2z, d_u2z, sizeof(float)*(unsigned)(imageSize), hipMemcpyDeviceToHost);
    
    mexPrintf("number of iterations = %i\n",punum[0]);
    
    
    /* Free memory */
    free( (float *) h_b1x1 );
    free( (float *) h_b1x2 );
    free( (float *) h_b1x3 );
    free( (float *) h_b1y1 );
    free( (float *) h_b1y2 );
    free( (float *) h_b1y3 );
    free( (float *) h_b1z1 );
    free( (float *) h_b1z2 );
    free( (float *) h_b1z3 );
    free( (float *) h_gkx );
    free( (float *) h_gky );
    free( (float *) h_gkz );
    free( (float *) h_dv1x );
    free( (float *) h_dv1y );
    free( (float *) h_dv1z );
    free( (float *) h_q1 );

    free( (float *) h_b2x1 );
    free( (float *) h_b2x2 );
    free( (float *) h_b2x3 );
    free( (float *) h_b2y1 );
    free( (float *) h_b2y2 );
    free( (float *) h_b2y3 );
    free( (float *) h_b2z1 );
    free( (float *) h_b2z2 );
    free( (float *) h_b2z3 );
    free( (float *) h_dv2x );
    free( (float *) h_dv2y );
    free( (float *) h_dv2z );
    free( (float *) h_q2 );
    
    free( (float *) h_FPS );
    free( (float *) h_qq1 );
    free( (float *) h_qq2 );
    free( (float *) h_qq3 );
    
    /*    Free GPU Memory */
    hipFree(d_b1x1);
    hipFree(d_b1x2);
    hipFree(d_b1x3);
    hipFree(d_b1y1);
    hipFree(d_b1y2);
    hipFree(d_b1y3);
    hipFree(d_b1z1);
    hipFree(d_b1z2);
    hipFree(d_b1z3);
    hipFree(d_gkx);
    hipFree(d_gky);
    hipFree(d_gkz);
    hipFree(d_dv1x);
    hipFree(d_dv1y);
    hipFree(d_dv1z);
    
    hipFree(d_u1x);
    hipFree(d_u1y);
    hipFree(d_u1z);
    hipFree(d_U1x);
    hipFree(d_U1y);
    hipFree(d_U1z);
    hipFree(d_G1x);
    hipFree(d_G1y);
    hipFree(d_G1z);
    hipFree(d_G1t);
    hipFree(d_G1f);
    hipFree(d_q1);

    hipFree(d_b2x1);
    hipFree(d_b2x2);
    hipFree(d_b2x3);
    hipFree(d_b2y1);
    hipFree(d_b2y2);
    hipFree(d_b2y3);
    hipFree(d_b2z1);
    hipFree(d_b2z2);
    hipFree(d_b2z3);
    hipFree(d_dv2x);
    hipFree(d_dv2y);
    hipFree(d_dv2z);
    
    hipFree(d_u2x);
    hipFree(d_u2y);
    hipFree(d_u2z);
    hipFree(d_U2x);
    hipFree(d_U2y);
    hipFree(d_U2z);
    hipFree(d_G2x);
    hipFree(d_G2y);
    hipFree(d_G2z);
    hipFree(d_G2t);
    hipFree(d_G2f);
    hipFree(d_q2);

    hipFree(d_FPS);
    hipFree(d_qq1);
    hipFree(d_qq2);
    hipFree(d_qq3);

    /* CUDA event-based timer */
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    hipEventDestroy( start );
    hipEventDestroy( stop );
    
    
    tt[0] = time;
    
    mexPrintf("\nComputational Time for Dual Optimization = %.4f sec\n \n",tt[0]/1000000);
    
    
}
