#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mex.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "CoSeg_kernels.cu"

#define YES 0
#define NO 1

#define PI 3.1415926

#define MAX(a,b) ( a > b ? a : b )
#define MIN(a,b) ( a <= b ? a : b )
#define SIGN(x) ( x >= 0.0 ? 1.0 : -1.0 )
#define ABS(x) ( (x) > 0.0 ? x : -(x) )

#ifndef HAVE_RINT 
#define rint(A) floor((A)+(((A) < 0)? -0.5 : 0.5)) 
#endif


/**********************************************/
/************** MAIN FUNCTION *****************/
/**********************************************/

/****************************************/
extern void mexFunction(int iNbOut, mxArray *pmxOut[],
int iNbIn, const mxArray *pmxIn[])
{
    
  /* iNbOut: number of outputs
     pmxOut: array of pointers to output arguments */
    
  /* iNbIn: number of inputs
     pmxIn: array of pointers to input arguments */
    
    
    float   *pfpenalty1, *pfpenalty2, *pfu1, *pfu2, *pfCs1, *pfCs2;
    float   *pfCt1, *pfCt2, *pfqq, *pfcvg, *pfVecParameters;
    float   *pfbx1, *pfbx2, *pfby1, *pfby2, *pfbz1, *pfbz2, *pfps1, *pfps2;
    float   *pfpt1, *pfpt2, *pfgk1, *pfgk2, *tt, *pfdv1, *pfdv2;

    float   fError, cc, steps, fps, beta;
    int     *punum, iNy, iNx, iNz, iNdim, iDim[3], ix, iy, iNI;
    int     SZF, idz, iz;

    int iDev;
    int     iNbIters, szImg, idx, index;
    time_t  start_time, end_time;

    //    GPU Variables
    float   *pfbx1_GPU, *pfby1_GPU, *pfpenalty1_GPU, *pfdv1_GPU;
    float   *pfbx2_GPU, *pfby2_GPU, *pfpenalty2_GPU, *pfdv2_GPU;
    float   *pfbz1_GPU, *pfbz2_GPU;
    float   *pfps1_GPU, *pfpt1_GPU, *pfgk1_GPU, *pfu1_GPU, *pfCs1_GPU, *pfCt1_GPU;
    float   *pfps2_GPU, *pfpt2_GPU, *pfgk2_GPU, *pfu2_GPU, *pfCs2_GPU, *pfCt2_GPU;
    float   *FPS, *FPS_GPU, *pfqq_GPU;
    
    
    hipDeviceProp_t prop;

    hipGetDeviceCount(&iDev);

    if ((unsigned int)iDev == 0){
        printf("There is no CUDA device found!");
        return;
    }
    else{
        printf("There are %d CUDA devices in your computer. \n", iDev);
        for(int ii = 0; ii < iDev; ii ++){
            hipGetDeviceProperties(&prop, ii);
            printf("------ General Information for CUDA device %d ------ \n", ii);
            printf("Name:  %s \n", prop.name);
            printf("Multiprocessor count:  %d \n", prop.multiProcessorCount);
            printf("Total global memory: %ld \n", prop.totalGlobalMem);
            printf("---------------------------------------------------- \n\n");
         }
    }

    /* Inputs */
    pfpenalty1 = (float*)mxGetData(pmxIn[0]); /* Given penalty1,2 */
    pfpenalty2 = (float*)mxGetData(pmxIn[1]);
    pfCs1 = (float*)mxGetData(pmxIn[2]); /* bound of source flows ps1 and ps2 */
    pfCs2 = (float*)mxGetData(pmxIn[3]);
    pfCt1 = (float*)mxGetData(pmxIn[4]); /* bound of sink flows pt1 and pt2 */
    pfCt2 = (float*)mxGetData(pmxIn[5]);
    pfVecParameters = (float*)mxGetData(pmxIn[6]); /* Vector of parameters */

     /* 
     *pfVecParameters Setting
     * [0] : number of columns 
     * [1] : number of rows
     * [2] : the maximum iteration number
     * [3] : error criterion
     * [4] : cc for the step-size of ALM
     * [5] : steps for the step-size of projected-gradient of p
     */
   
    /* Size */
    iNy = (int) pfVecParameters[0];
    iNx = (int) pfVecParameters[1];
    iNz = (int) pfVecParameters[2];
    szImg = iNy*iNx*iNz;
    SZF = iNy*iNx;

    /* Choice of region segmentation model */
    iNbIters = (int) pfVecParameters[3]; /* the maximum iteration number */
    fError = (float) pfVecParameters[4]; /* error bound for convergence */
    cc = (float) pfVecParameters[5]; /* the step-size for ALM */
    steps = (float) pfVecParameters[6]; /* the step-size for each projected-gradient step */
    beta = (float) pfVecParameters[7];

    printf("Initializing ................................................ \n\n");

    /* Outputs */
    /* outputs the computed u1(x)  */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    pmxOut[0] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    pfu1 = (float*)mxGetData(pmxOut[0]);
    
    /* outputs the computed u2(x)  */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;

    pmxOut[1] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    pfu2 = (float*)mxGetData(pmxOut[1]);
   
    /* outputs the convergence rate  */
    iNdim = 2;
    iDim[0] = 1;
    iDim[1] = iNbIters;
    
    pmxOut[2] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    pfcvg = (float*)mxGetData(pmxOut[2]);
    
    /* outputs the iteration number  */
    iNdim = 2;
    iDim[0] = 1;
    iDim[1] = 1;
    pmxOut[3] = mxCreateNumericArray(iNdim,(const int*)iDim,mxUINT16_CLASS,mxREAL);
    punum = (int*)mxGetData(pmxOut[3]);
    
    /* outputs the computation time  */
    iNdim = 2;
    iDim[0] = 1;
    iDim[1] = 1;
    pmxOut[4] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    tt = (float*)mxGetData(pmxOut[4]);
    
    /* Memory allocation */
    
    /* allocate the memory for px1 and px2 */
    pfbx1 = (float *) calloc( (unsigned)(iNy*(iNx+1)*iNz), sizeof(float) );
    if (!pfbx1)
        mexPrintf("Memory allocation failure\n");
    
    pfbx2 = (float *) calloc( (unsigned)(iNy*(iNx+1)*iNz), sizeof(float) );
    if (!pfbx2)
        mexPrintf("Memory allocation failure\n");
    
    /* allocate the memory for py1 and py2 */
    pfby1 = (float *) calloc( (unsigned)((iNy+1)*iNx*iNz), sizeof(float) );
    if (!pfby1)
        mexPrintf("Memory allocation failure\n");
    
    pfby2 = (float *) calloc( (unsigned)((iNy+1)*iNx*iNz), sizeof(float) );
    if (!pfby2)
        mexPrintf("Memory allocation failure\n");
    
    /* allocate the memory for pz1 and pz2 */
    pfbz1 = (float *) calloc( (unsigned)(iNy*iNx*(iNz+1)), sizeof(float) );
    if (!pfbz1)
        mexPrintf("Memory allocation failure\n");

    pfbz2 = (float *) calloc( (unsigned)(iNy*iNx*(iNz+1)), sizeof(float) );
    if (!pfbz2)
        mexPrintf("Memory allocation failure\n");

    /* allocate the memory for ps1 and ps2 */
    pfps1 = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!pfps1)
        mexPrintf("Memory allocation failure\n");
    
    pfps2 = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!pfps2)
        mexPrintf("Memory allocation failure\n");
    
    /* allocate the memory for pt1 and pt2 */
    pfpt1 = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!pfpt1)
        mexPrintf("Memory allocation failure\n");
    
    pfpt2 = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!pfpt2)
        mexPrintf("Memory allocation failure\n");
    
    /* allocate the memory for the coupled flow q */
    pfqq = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!pfqq)
        mexPrintf("Memory allocation failure\n");
    
    /* allocate the memory for gk1 */
    pfgk1 = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!pfgk1)
        mexPrintf("Memory allocation failure\n");
    
    /* allocate the memory for gk2 */
    pfgk2 = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!pfgk2)
        mexPrintf("Memory allocation failure\n");
    
    /* allocate the memory for div1 and div2 */
    pfdv1 = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!pfdv1)
        mexPrintf("Memory allocation failure\n");
    
    pfdv2 = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!pfdv2)
        mexPrintf("Memory allocation failure\n");

    /* allocate the memory for FPS */
    FPS = (float *) calloc( (unsigned)(iNy*iNx*iNz), sizeof(float) );
    if (!FPS)
        mexPrintf("Memory allocation failure\n");


     //    GPU Memory Allocation
    
    hipMalloc( (void**) &pfbx1_GPU, sizeof(float)*(unsigned)((iNx+1)*iNy*iNz));
    hipMalloc( (void**) &pfby1_GPU, sizeof(float)*(unsigned)(iNx*(iNy+1)*iNz));
    hipMalloc( (void**) &pfbz1_GPU, sizeof(float)*(unsigned)(iNx*iNy*(iNz+1)));
    hipMalloc( (void**) &pfpenalty1_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfdv1_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfps1_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfpt1_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfgk1_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfu1_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfCs1_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfCt1_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));

    hipMalloc( (void**) &pfbx2_GPU, sizeof(float)*(unsigned)((iNx+1)*iNy*iNz));
    hipMalloc( (void**) &pfby2_GPU, sizeof(float)*(unsigned)(iNx*(iNy+1)*iNz));
    hipMalloc( (void**) &pfbz2_GPU, sizeof(float)*(unsigned)(iNx*iNy*(iNz+1)));
    hipMalloc( (void**) &pfpenalty2_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfdv2_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfps2_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfpt2_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfgk2_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfu2_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfCs2_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &pfCt2_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));

    hipMalloc( (void**) &pfqq_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));
    hipMalloc( (void**) &FPS_GPU, sizeof(float)*(unsigned)(iNy*iNx*iNz));

    /* Preprocessing initial values */
    for (iz=0; iz < iNz; iz++){
        idz = iz*SZF;
        for (ix=0; ix< iNx; ix++){
            idx = idz + ix*iNy;
            for (iy=0; iy< iNy; iy++){
               index = idx + iy; 

                if (pfCs1[index] < pfCt1[index]){
                    pfps1[index] = pfCs1[index];
                    pfpt1[index] = pfCs1[index];
                    pfdv1[index] = pfbx1[index+iNy] - pfbx1[index] 
                                + pfby1[index+1] - pfby1[index]
                                + pfbz1[index+SZF] - pfbz1[index];
                }
                else{
                    pfu1[index] = 1;
                    pfps1[index] = pfCt1[index];
                    pfpt1[index] = pfCt1[index];
                    pfdv1[index] = pfbx1[index+iNy] - pfbx1[index] 
                                + pfby1[index+1] - pfby1[index]
                                + pfbz1[index+SZF] - pfbz1[index];
                }

               if (pfCs2[index] < pfCt2[index]){
                    pfps2[index] = pfCs2[index];
                    pfpt2[index] = pfCs2[index];
                    pfdv2[index] = pfbx2[index+iNy] - pfbx2[index] 
                                 + pfby2[index+1] - pfby2[index]
                                 + pfbz2[index+SZF] - pfbz2[index];
                }
                else{
                    pfu2[index] = 1;
                    pfps2[index] = pfCt2[index];
                    pfpt2[index] = pfCt2[index];
                    pfdv2[index] = pfbx2[index+iNy] - pfbx2[index] 
                                + pfby2[index+1] - pfby2[index]
                                + pfbz2[index+SZF] - pfbz2[index];
                }
            }
        }
    }
    
    //    Copy Parameters from Host to Device

    hipMemcpy( pfbx1_GPU, pfbx1, sizeof(float)*(unsigned)(iNy*(iNx+1)*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfby1_GPU, pfby1, sizeof(float)*(unsigned)((iNy+1)*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfbz1_GPU, pfbz1, sizeof(float)*(unsigned)(iNy*iNx*(iNz+1)), hipMemcpyHostToDevice);
    hipMemcpy( pfpenalty1_GPU, pfpenalty1, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfdv1_GPU, pfdv1, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfps1_GPU, pfps1, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfpt1_GPU, pfpt1, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfgk1_GPU, pfgk1, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfu1_GPU, pfu1, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfCs1_GPU, pfCs1, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfCt1_GPU, pfCt1, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);

    hipMemcpy( pfbx2_GPU, pfbx2, sizeof(float)*(unsigned)(iNy*(iNx+1)*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfby2_GPU, pfby2, sizeof(float)*(unsigned)((iNy+1)*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfbz2_GPU, pfbz2, sizeof(float)*(unsigned)(iNy*iNx*(iNz+1)), hipMemcpyHostToDevice);
    hipMemcpy( pfpenalty2_GPU, pfpenalty2, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfdv2_GPU, pfdv2, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfps2_GPU, pfps2, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfpt2_GPU, pfpt2, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfgk2_GPU, pfgk2, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfu2_GPU, pfu2, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfCs2_GPU, pfCs2, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( pfCt2_GPU, pfCt2, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    
    hipMemcpy( pfqq_GPU, pfqq, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);
    hipMemcpy( FPS_GPU, FPS, sizeof(float)*(unsigned)(iNy*iNx*iNz), hipMemcpyHostToDevice);

    /*  Main iterations */
    
    iNI = 0;

    /* Compute the execution configuration */

    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,BLOCK_SIZE);
  
    int blocksInX = (iNy/dimBlock.x) + (!(iNy%dimBlock.x)?0:1);
    int blocksInY = (iNx/dimBlock.y) + (!(iNx%dimBlock.y)?0:1);
    int blocksInZ = (iNz/dimBlock.z) + (!(iNz%dimBlock.z)?0:1);

    dim3 dimGrid ( blocksInX, blocksInY*blocksInZ);

    blocksInX = ((iNy-1)/dimBlock.x) + (!((iNy-1)%dimBlock.x)?0:1);
    int blocksInY_x = (iNx/dimBlock.y) + (!(iNx%dimBlock.y)?0:1);
    blocksInZ = (iNz/dimBlock.z) + (!(iNz%dimBlock.z)?0:1);

    dim3 dimGrid_x (blocksInX, blocksInY_x*blocksInZ);

    blocksInX = (iNy/dimBlock.x) + (!(iNy%dimBlock.x)?0:1);
    int blocksInY_y = ((iNx-1)/dimBlock.y) + (!((iNx-1)%dimBlock.y)?0:1);
    blocksInZ = (iNz/dimBlock.z) + (!(iNz%dimBlock.z)?0:1);

    dim3 dimGrid_y ( blocksInX, blocksInY_y*blocksInZ);

    blocksInX = (iNy/dimBlock.x) + (!(iNy%dimBlock.x)?0:1);
    int blocksInY_z = (iNx/dimBlock.y) + (!(iNx%dimBlock.y)?0:1);
    blocksInZ = ((iNz-1)/dimBlock.z) + (!((iNz-1)%dimBlock.z)?0:1);
 
    dim3 dimGrid_z (blocksInX, blocksInY_z*blocksInZ);
    
    start_time = clock();

    printf("Start computing ......................................... \n\n");

    while( iNI<iNbIters ) 
    { 

        /* update px */
        krnl_1<<< dimGrid, dimBlock>>>(pfpt1_GPU, pfps1_GPU, pfu1_GPU, 
                    pfgk1_GPU, pfdv1_GPU, pfpt2_GPU, pfps2_GPU, pfu2_GPU, 
                    pfgk2_GPU, pfdv2_GPU, pfqq_GPU, cc, iNx, iNy, iNz, SZF, 
                    blocksInY, 1.0f/(float)blocksInY);
       
        krnl_2<<< dimGrid_y, dimBlock>>>(pfbx1_GPU, pfgk1_GPU, pfbx2_GPU, 
                    pfgk2_GPU, steps, iNx, iNy, iNz, SZF, blocksInY_y, 1.0f/(float)blocksInY_y);

        krnl_3<<< dimGrid_x, dimBlock>>>(pfby1_GPU, pfgk1_GPU, pfby2_GPU, 
                    pfgk2_GPU, steps, iNx, iNy, iNz, SZF, blocksInY_x, 1.0f/(float)blocksInY_x);
      
        krnl_z<<<dimGrid_z, dimBlock>>>(pfbz1_GPU, pfgk1_GPU, pfbz2_GPU, pfgk2_GPU, steps, 
                  iNx, iNy, iNz, SZF, blocksInY_z, 1.0f/(float)blocksInY_z);

        /* projection step */
        krnl_4<<< dimGrid, dimBlock>>>(pfbx1_GPU, pfby1_GPU, pfbz1_GPU, pfgk1_GPU, pfpenalty1_GPU, 
                    pfbx2_GPU, pfby2_GPU, pfbz2_GPU, pfgk2_GPU, pfpenalty2_GPU, iNx, iNy,
                    iNz, SZF, blocksInY, 1.0f/(float)blocksInY);
      
        krnl_5<<< dimGrid_y, dimBlock >>>(pfbx1_GPU, pfgk1_GPU, pfbx2_GPU, 
                    pfgk2_GPU, iNx, iNy, iNz, SZF, blocksInY_y, 1.0f/(float)blocksInY_y);
    
        krnl_6<<< dimGrid_x, dimBlock >>>(pfby1_GPU, pfgk1_GPU, pfby2_GPU, 
                    pfgk2_GPU, iNx, iNy, iNz, SZF, blocksInY_x, 1.0f/(float)blocksInY_x);

        krnl_zp<<<dimGrid_z, dimBlock>>>(pfbz1_GPU, pfgk1_GPU, pfbz2_GPU, pfgk2_GPU, iNx, 
                    iNy, iNz, SZF, blocksInY_z, 1.0f/(float)blocksInY_z);

        /* compute the divergence  */
        krnl_7<<< dimGrid, dimBlock>>>(pfbx1_GPU, pfby1_GPU, pfbz1_GPU, pfdv1_GPU, 
                    pfbx2_GPU, pfby2_GPU, pfbz2_GPU, pfdv2_GPU, iNx, iNy, iNz, SZF, 
                    blocksInY, 1.0f/(float)blocksInY);

        /* update ps  */
        krnl_8<<< dimGrid, dimBlock>>>(pfps1_GPU, pfpt1_GPU, pfu1_GPU, pfdv1_GPU, 
                    pfCs1_GPU, pfps2_GPU, pfpt2_GPU, pfu2_GPU, pfdv2_GPU, pfCs2_GPU, 
                    pfqq_GPU, cc, iNx, iNy, iNz, SZF, blocksInY, 1.0f/(float)blocksInY);
        
        /* update pt  */
        krnl_9<<< dimGrid, dimBlock>>>(pfps1_GPU, pfpt1_GPU, pfu1_GPU, pfdv1_GPU, 
                    pfCt1_GPU, pfps2_GPU, pfpt2_GPU, pfu2_GPU, pfdv2_GPU, 
                    pfCt2_GPU, pfqq_GPU, cc, iNx, iNy, iNz, SZF, blocksInY, 1.0f/(float)blocksInY);
   
        /* update qq  */
        krnl_qq<<< dimGrid, dimBlock>>>(pfps1_GPU, pfpt1_GPU, pfu1_GPU, pfdv1_GPU, 
                    pfps2_GPU, pfpt2_GPU, pfu2_GPU, pfdv2_GPU, 
                    pfqq_GPU, beta, cc, iNx, iNy, iNz, SZF, blocksInY, 1.0f/(float)blocksInY);

        /* update multipliers */
        krnl_10<<< dimGrid, dimBlock>>>(pfpt1_GPU, pfdv1_GPU, pfps1_GPU, pfu1_GPU, 
                pfpt2_GPU, pfdv2_GPU, pfps2_GPU, pfu2_GPU, pfqq_GPU,
                FPS_GPU, cc, iNx, iNy, iNz, SZF, blocksInY, 1.0f/(float)blocksInY);

        hipMemcpy( FPS, FPS_GPU, sizeof(float)*(unsigned)(szImg), hipMemcpyDeviceToHost);

        fps = 0;
        for (int ii=0; ii< szImg; ii++){
                fps += FPS[ii];
        }

        pfcvg[iNI] = fps / szImg / 2;
        
        if (pfcvg[iNI] <= fError){
            break;
        }

        iNI ++;
     }   

    hipMemcpy( pfu1, pfu1_GPU, sizeof(float)*(unsigned)(szImg), hipMemcpyDeviceToHost);
    hipMemcpy( pfu2, pfu2_GPU, sizeof(float)*(unsigned)(szImg), hipMemcpyDeviceToHost);

    mexPrintf("Total iteration number = %i\n",iNI);
    end_time = clock();

    /* Outputs (see above) */
    punum[0] = iNI;
    
    /* Free memory */
    free( (float *) pfbx1 );
    free( (float *) pfby1 );
    free( (float *) pfbz1 );
    free( (float *) pfps1 );
    free( (float *) pfpt1 );
    free( (float *) pfgk1 );
    free( (float *) pfdv1 );

    free( (float *) pfbx2 );
    free( (float *) pfby2 );
    free( (float *) pfbz2 );
    free( (float *) pfps2 );
    free( (float *) pfpt2 );
    free( (float *) pfgk2 );
    free( (float *) pfdv2 );

    free( (float *) pfqq );
    free( (float *) FPS );

    //    Free GPU Memory
    hipFree(pfbx1_GPU);
    hipFree(pfby1_GPU);
    hipFree(pfbz1_GPU);
    hipFree(pfpenalty1_GPU);
    hipFree(pfps1_GPU);
    hipFree(pfpt1_GPU);
    hipFree(pfgk1_GPU);
    hipFree(pfdv1_GPU);
    hipFree(pfu1_GPU);
    hipFree(pfCs1_GPU);
    hipFree(pfCt1_GPU);

    hipFree(pfbx2_GPU);
    hipFree(pfby2_GPU);
    hipFree(pfbz2_GPU);
    hipFree(pfpenalty2_GPU);
    hipFree(pfps2_GPU);
    hipFree(pfpt2_GPU);
    hipFree(pfgk2_GPU);
    hipFree(pfdv2_GPU);
    hipFree(pfu2_GPU);
    hipFree(pfCs2_GPU);
    hipFree(pfCt2_GPU);

    hipFree(pfqq_GPU);
    hipFree(FPS_GPU);

    tt[0] = difftime(end_time,start_time)/1000000;
    mexPrintf("\nComputing Time for max-flow = %.4f sec\n \n",tt[0]);
    
    
}
/****************************************/






/**********************************************/
/************** END MAIN FUNCTION *************/
/**********************************************/
