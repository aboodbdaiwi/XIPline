
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 8
#define MAX(a,b) ( a > b ? a : b )
#define MIN(a,b) ( a <= b ? a : b )
#define SIGN(x) ( x >= 0.0 ? 1.0 : -1.0 )
#define ABS(x) ( (x) > 0.0 ? x : -(x) )
#define SQR(x) (x)*(x)

static __global__ void krnl_1(float *pfpt1, float *pfps1, float *pfu1, 
        float *pfgk1, float *pfdv1, float *pfpt2, float *pfps2, float *pfu2, 
        float *pfgk2, float *pfdv2, float *pfqq, float cc, int iNx, int iNy,
        int iNz, int SZF, int blocksInY, float invBlocksInY)
{
   int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
   int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
   int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
   int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
   int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;

   if( idx<iNy && idy<iNx && idz<iNz)
   {
    int index = idx + __mul24(idy, iNy) + __mul24(idz, SZF);
    
    pfgk1[index] = pfdv1[index] - (pfps1[index] - pfpt1[index] 
                + pfu1[index]/cc + pfqq[index]);

    pfgk2[index] = pfdv2[index] - (pfps2[index] - pfpt2[index] 
                + pfu2[index]/cc - pfqq[index]);
   }

}

static __global__ void krnl_2(float *pfbx1, float *pfgk1, float *pfbx2, 
            float *pfgk2, float steps, int iNx, int iNy,
            int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;

    if( idx<iNy && idy<(iNx-1) && idz<iNz)
    {
      int index = idx + __mul24(idy+1, iNy) + __mul24(idz, SZF);

      pfbx1[index] = steps*(pfgk1[index] - pfgk1[index-iNy]) + pfbx1[index];
      pfbx2[index] = steps*(pfgk2[index] - pfgk2[index-iNy]) + pfbx2[index];  
    }
}

static __global__ void krnl_3(float *pfby1, float *pfgk1, float *pfby2, float *pfgk2,
            float steps, int iNx, int iNy, int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;
    
    if( idx<(iNy-1) && idy<iNx && idz<iNz)
    {
      int index =idx + __mul24(idy, iNy) + __mul24(idz, SZF) + 1;
      
      pfby1[index] = steps*(pfgk1[index] - pfgk1[index-1]) + pfby1[index];
      pfby2[index] = steps*(pfgk2[index] - pfgk2[index-1]) + pfby2[index];
    }
}

static __global__ void krnl_z(float *pfbz1, float *pfgk1, float *pfbz2, float *pfgk2, 
        float steps, int iNx, int iNy, int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;
    
    if( idx<iNy && idy<iNx && idz<(iNz-1))
    {
      int index = idx + __mul24(idy, iNy) + __mul24(idz+1, SZF);
    
      pfbz1[index] = __fadd_rz(__fmul_rz(steps, __fadd_rz(pfgk1[index], - pfgk1[index-SZF])), pfbz1[index]);
      pfbz2[index] = __fadd_rz(__fmul_rz(steps, __fadd_rz(pfgk2[index], - pfgk2[index-SZF])), pfbz2[index]);
    }
}

static __global__ void krnl_4(float *pfbx1, float *pfby1, float *pfbz1, float *pfgk1, 
        float *pfpenalty1, float *pfbx2, float *pfby2, float *pfbz2, float *pfgk2, 
        float *pfpenalty2, int iNx, int iNy, int iNz, int SZF, int blocksInY, 
        float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;
    float fpt;
   
    if( idx<iNy && idy<iNx && idz<iNz)
    {
      int index = idx + __mul24(idy, iNy) + __mul24(idz, SZF);

      fpt = sqrt((SQR(pfbx1[index]) + SQR(pfbx1[index+iNy]) 
            + SQR(pfby1[index]) + SQR(pfby1[index+1]) +
            + SQR(pfbz1[index])+ SQR(pfbz1[index+SZF]))*0.5);
                
      if (fpt > pfpenalty1[index])
          pfgk1[index] = pfpenalty1[index]/fpt;
      else
          pfgk1[index] = 1;

      fpt = sqrt((SQR(pfbx2[index]) + SQR(pfbx2[index+iNy]) 
            + SQR(pfby2[index]) + SQR(pfby2[index+1])
            + SQR(pfbz2[index])+ SQR(pfbz2[index+SZF]))*0.5);
                
      if (fpt > pfpenalty2[index])
          pfgk2[index] = pfpenalty2[index]/fpt;
      else
          pfgk2[index] = 1;

    }
}

static __global__ void krnl_5(float *pfbx1, float *pfgk1, float *pfbx2, 
            float *pfgk2, int iNx, int iNy, int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;

    if( idx<iNy && idy<(iNx-1) && idz<iNz)
    {
      int index = idx + __mul24(idy+1, iNy) + __mul24(idz, SZF);  
       
      pfbx1[index] = (pfgk1[index] + pfgk1[index-iNy])*0.5*pfbx1[index];
      pfbx2[index] = (pfgk2[index] + pfgk2[index-iNy])*0.5*pfbx2[index];
    }
}


static __global__ void krnl_6(float *pfby1, float *pfgk1, float *pfby2, 
            float *pfgk2, int iNx, int iNy, int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;
    
    if( idx<(iNy-1) && idy<iNx && idz<iNz)
    {
      int index = idx + __mul24(idy, iNy) + __mul24(idz, SZF)+1;
      
      pfby1[index] = 0.5*(pfgk1[index] + pfgk1[index-1])*pfby1[index];
      pfby2[index] = 0.5*(pfgk2[index-1] + pfgk2[index])*pfby2[index];
    }
}

static __global__ void krnl_zp(float *pfbz1, float *pfgk1, float *pfbz2, float *pfgk2, int iNx, 
        int iNy, int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;
    
    if( idx<iNy && idy<iNx && idz<(iNz-1))
    {
      int index = idx + __mul24(idy, iNy) + __mul24(idz+1, SZF);

      pfbz1[index] = __fmul_rz(__fmul_rz(__fadd_rz(pfgk1[index], pfgk1[index-SZF]), 0.5), pfbz1[index]);
      pfbz2[index] = __fmul_rz(__fmul_rz(__fadd_rz(pfgk2[index], pfgk2[index-SZF]), 0.5), pfbz2[index]);
    }
}

static __global__ void krnl_7(float *pfbx1, float *pfby1, float *pfbz1, float *pfdv1, 
                  float *pfbx2, float *pfby2, float *pfbz2, float *pfdv2, int iNx, int iNy,
                  int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;

   if( idx<iNy && idy<iNx && idz<iNz)
   {
     int index = idx + __mul24(idy, iNy) + __mul24(idz, SZF);

     pfdv1[index] = pfbx1[index+iNy] - pfbx1[index] 
                + pfby1[index+1] - pfby1[index]
                + pfbz1[index+SZF] - pfbz1[index];

     pfdv2[index] = pfbx2[index+iNy] - pfbx2[index] 
                + pfby2[index+1] - pfby2[index]
                + pfbz2[index+SZF] - pfbz2[index];
    }
}
 

static __global__ void krnl_8(float *pfps1, float *pfpt1, float *pfu1, float *pfdv1, 
                float *pfCs1, float *pfps2, float *pfpt2, float *pfu2, float *pfdv2, 
                float *pfCs2, float *pfqq, float cc, int iNx, int iNy,
                int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;
    float fpt;
    
    if( idx<iNy && idy<iNx && idz<iNz)
    {
      int index = idx + __mul24(idy, iNy) + __mul24(idz, SZF);

      fpt = pfpt1[index] - pfu1[index]/cc + pfdv1[index] - pfqq[index] + 1/cc;
      pfps1[index] = MIN(fpt, pfCs1[index]);

      fpt = pfpt2[index] - pfu2[index]/cc + pfdv2[index] + pfqq[index] + 1/cc;
      pfps2[index] = MIN(fpt , pfCs2[index]);
    }
}


static __global__ void krnl_9(float *pfps1, float *pfpt1, float *pfu1, float *pfdv1, 
                    float *pfCt1, float *pfps2, float *pfpt2, float *pfu2, float *pfdv2, 
                    float *pfCt2, float *pfqq, float cc, int iNx, int iNy,
                    int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;
    float fpt;
    
    if( idx<iNy && idy<iNx && idz<iNz)
    {
      int index = idx + __mul24(idy, iNy) + __mul24(idz, SZF);

      fpt = pfps1[index] + pfu1[index]/cc - pfdv1[index] + pfqq[index];
      pfpt1[index] = MIN(fpt, pfCt1[index]);
        
      fpt = pfps2[index] + pfu2[index]/cc - pfdv2[index] - pfqq[index];
      pfpt2[index] = MIN(fpt , pfCt2[index]);
    }
}

static __global__ void krnl_qq(float *pfps1, float *pfpt1, float *pfu1, float *pfdv1, 
                    float *pfps2, float *pfpt2, float *pfu2, float *pfdv2, 
                    float *pfqq, float beta, float cc, int iNx, int iNy,
                    int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;
    float fpt;
    
    if( idx<iNy && idy<iNx && idz<iNz)
    {
      int index = idx + __mul24(idy, iNy) + __mul24(idz, SZF);

      fpt = ((pfdv1[index] - pfps1[index] - pfu1[index]/cc + pfpt1[index]) +
           (pfps2[index] + pfu2[index]/cc - pfdv2[index] - pfpt2[index]))/2;
      pfqq[index] = MAX(MIN(fpt , beta), -beta);
    }
}

static __global__ void krnl_10(float *pfpt1, float *pfdv1, float *pfps1, float *pfu1,
                    float *pfpt2, float *pfdv2, float *pfps2, float *pfu2, float *pfqq,
                    float *FPS, float cc, int iNx, int iNy, 
                    int iNz, int SZF, int blocksInY, float invBlocksInY)
{
    int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
    int blockIdxy = blockIdx.y - __umul24(blockIdxz,blocksInY);
    int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
    int idy   = __mul24(blockIdxy,blockDim.y)+threadIdx.y;
    int idz   = __mul24(blockIdxz,blockDim.z)+threadIdx.z;
    float fpt;

    if( idx<iNy && idy<iNx && idz<iNz)
    {
      int index = idx + __mul24(idy, iNy) + __mul24(idz, SZF);

      fpt = cc*(pfpt1[index] + pfdv1[index] - pfps1[index] - pfqq[index]);
      FPS[index] = ABS(fpt);

      pfu1[index] -= fpt;

      fpt = cc*(pfpt2[index] + pfdv2[index] - pfps2[index] + pfqq[index]);
      FPS[index] += ABS(fpt);

      pfu2[index] -= fpt;
    }
}