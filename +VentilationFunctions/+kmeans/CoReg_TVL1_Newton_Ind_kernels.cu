
#include <hip/hip_runtime.h>
#define MAX(a,b) ( a > b ? a : b )
#define MIN(a,b) ( a <= b ? a : b )

static __global__ void krnl_1(float *dvx, float *dvy, float *dvz,
        float *ux, float *uy, float *uz,
        float *gkx, float *gky, float *gkz,
        float *Gx, float *Gy, float *Gz,
        float *Gt, float *Gf, float *q,
        float *Ux, float *Uy, float *Uz,
        float cc, int iNx, int iNy, int iNz, float *d_qq1, float *d_qq2, float *d_qq3, int flag, float weight){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;

    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        float tx = dvx[idx] + flag*d_qq1[idx] - ux[idx]/cc;
        float ty = dvy[idx] + flag*d_qq2[idx] - uy[idx]/cc;
        float tz = dvz[idx] + flag*d_qq3[idx] - uz[idx]/cc;
        
        q[idx] = (Gt[idx] - cc*(Gx[idx]*tx + Gy[idx]*ty + Gz[idx]*tz))/(1 + cc*Gf[idx]);
        q[idx] = MAX(MIN(q[idx],weight),-weight);     /*this was added by Fumin Guo, 2015/08/28*/  
        
        gkx[idx] = tx + q[idx]*Gx[idx] - Ux[idx]/cc*2;
        gky[idx] = ty + q[idx]*Gy[idx] - Uy[idx]/cc*2;
        gkz[idx] = tz + q[idx]*Gz[idx] - Uz[idx]/cc*2;
        
    }
}

static __global__ void krnl_2(float *bx1, float *by1, float *bz1,
        float *gkx, float *gky, float *gkz,
        float steps, int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        bx1[idx+iNy] = steps*(gkx[idx+iNy] - gkx[idx]) + bx1[idx+iNy];
        by1[idx+iNy] = steps*(gky[idx+iNy] - gky[idx]) + by1[idx+iNy];
        bz1[idx+iNy] = steps*(gkz[idx+iNy] - gkz[idx]) + bz1[idx+iNy];
    }
}

static __global__ void krnl_3(float *bx2, float *by2, float *bz2,
        float *gkx, float *gky, float *gkz,
        float steps, int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        bx2[idx+1] = steps*(gkx[idx+1] - gkx[idx]) + bx2[idx+1];
        by2[idx+1] = steps*(gky[idx+1] - gky[idx]) + by2[idx+1];
        bz2[idx+1] = steps*(gkz[idx+1] - gkz[idx]) + bz2[idx+1];
        
    }
}

static __global__ void krnl_z(float *bx3, float *by3, float *bz3,
        float *gkx, float *gky, float *gkz,
        float steps, int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        bx3[idx+(iNx*iNy)] = steps*(gkx[idx+(iNx*iNy)] - gkx[idx]) + bx3[idx+(iNx*iNy)];
        by3[idx+(iNx*iNy)] = steps*(gky[idx+(iNx*iNy)] - gky[idx]) + by3[idx+(iNx*iNy)];
        bz3[idx+(iNx*iNy)] = steps*(gkz[idx+(iNx*iNy)] - gkz[idx]) + bz3[idx+(iNx*iNy)];
    }
}

static __global__ void krnl_23z(float *bx1, float *by1, float *bz1,
        float *bx2, float *by2, float *bz2,
        float *bx3, float *by3, float *bz3,
        float *gkx, float *gky, float *gkz,
        float steps, int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        bx1[idx+iNy] = steps*(gkx[idx+iNy] - gkx[idx]) + bx1[idx+iNy];
        by1[idx+iNy] = steps*(gky[idx+iNy] - gky[idx]) + by1[idx+iNy];
        bz1[idx+iNy] = steps*(gkz[idx+iNy] - gkz[idx]) + bz1[idx+iNy];
        
        bx2[idx+1] = steps*(gkx[idx+1] - gkx[idx]) + bx2[idx+1];
        by2[idx+1] = steps*(gky[idx+1] - gky[idx]) + by2[idx+1];
        bz2[idx+1] = steps*(gkz[idx+1] - gkz[idx]) + bz2[idx+1];
        
        bx3[idx+(iNx*iNy)] = steps*(gkx[idx+(iNx*iNy)] - gkx[idx]) + bx3[idx+(iNx*iNy)];
        by3[idx+(iNx*iNy)] = steps*(gky[idx+(iNx*iNy)] - gky[idx]) + by3[idx+(iNx*iNy)];
        bz3[idx+(iNx*iNy)] = steps*(gkz[idx+(iNx*iNy)] - gkz[idx]) + bz3[idx+(iNx*iNy)];
        
    }
}

static __global__ void krnl_4(float *bx1, float *bx2, float *bx3,
        float *by1, float *by2, float *by3,
        float *bz1, float *bz2, float *bz3,
        float *gkx, float *gky, float *gkz,
        float fPenalty, int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        
        
        float fpt = sqrtf((powf(bx1[idx],2) + powf(bx1[idx+iNy],2)
        + powf(bx2[idx],2) + powf(bx2[idx+1],2)
        + powf(bx3[idx],2) + powf(bx3[idx+(iNx*iNy)],2))*0.5);
        
        if (fpt > fPenalty)
            gkx[idx] = fPenalty/fpt;
        else
            gkx[idx] = 1;
        
        fpt = sqrtf((powf(by1[idx],2) + powf(by1[idx+iNy],2)
        + powf(by2[idx],2) + powf(by2[idx+1],2)
        + powf(by3[idx],2) + powf(by3[idx+(iNx*iNy)],2))*0.5);
        
        if (fpt > fPenalty)
            gky[idx] = fPenalty/fpt;
        else
            gky[idx] = 1;
        
        fpt = sqrtf((powf(bz1[idx],2) + powf(bz1[idx+iNy],2)
        + powf(bz2[idx],2) + powf(bz2[idx+1],2)
        + powf(bz3[idx],2) + powf(bz3[idx+(iNx*iNy)],2))*0.5);
        
        if (fpt > fPenalty)
            gkz[idx] = fPenalty/fpt;
        else
            gkz[idx] = 1;
        
    }
}

static __global__ void krnl_5(float *bx1, float *by1, float *bz1,
        float *gkx, float *gky, float *gkz,
        int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        
        bx1[idx+iNy] = (gkx[idx+iNy] + gkx[idx])*0.5*bx1[idx+iNy];
        by1[idx+iNy] = (gky[idx+iNy] + gky[idx])*0.5*by1[idx+iNy];
        bz1[idx+iNy] = (gkz[idx+iNy] + gkz[idx])*0.5*bz1[idx+iNy];
    }
}

static __global__ void krnl_6(float *bx2, float *by2, float *bz2,
        float *gkx, float *gky, float *gkz,
        int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        bx2[idx+1] = 0.5*(gkx[idx+1] + gkx[idx])*bx2[idx+1];
        by2[idx+1] = 0.5*(gky[idx+1] + gky[idx])*by2[idx+1];
        bz2[idx+1] = 0.5*(gkz[idx+1] + gkz[idx])*bz2[idx+1];
    }
}

static __global__ void krnl_zp(float *bx3, float *by3, float *bz3,
        float *gkx, float *gky, float *gkz,
        int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;

    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        bx3[idx+(iNx*iNy)] = 0.5*(gkx[idx+(iNx*iNy)] + gkx[idx])*bx3[idx+(iNx*iNy)];
        by3[idx+(iNx*iNy)] = 0.5*(gky[idx+(iNx*iNy)] + gky[idx])*by3[idx+(iNx*iNy)];
        bz3[idx+(iNx*iNy)] = 0.5*(gkz[idx+(iNx*iNy)] + gkz[idx])*bz3[idx+(iNx*iNy)];
    }
}

static __global__ void krnl_56zp(float *bx1, float *by1, float *bz1,
        float *bx2, float *by2, float *bz2,
        float *bx3, float *by3, float *bz3,
        float *gkx, float *gky, float *gkz,
        int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        bx1[idx+iNy] = (gkx[idx+iNy] + gkx[idx])*0.5*bx1[idx+iNy];
        by1[idx+iNy] = (gky[idx+iNy] + gky[idx])*0.5*by1[idx+iNy];
        bz1[idx+iNy] = (gkz[idx+iNy] + gkz[idx])*0.5*bz1[idx+iNy];
        
        bx2[idx+1] = 0.5*(gkx[idx+1] + gkx[idx])*bx2[idx+1];
        by2[idx+1] = 0.5*(gky[idx+1] + gky[idx])*by2[idx+1];
        bz2[idx+1] = 0.5*(gkz[idx+1] + gkz[idx])*bz2[idx+1];
        
        bx3[idx+(iNx*iNy)] = 0.5*(gkx[idx+(iNx*iNy)] + gkx[idx])*bx3[idx+(iNx*iNy)];
        by3[idx+(iNx*iNy)] = 0.5*(gky[idx+(iNx*iNy)] + gky[idx])*by3[idx+(iNx*iNy)];
        bz3[idx+(iNx*iNy)] = 0.5*(gkz[idx+(iNx*iNy)] + gkz[idx])*bz3[idx+(iNx*iNy)];
        
        
    }
}

static __global__ void krnl_7(float *bx1, float *bx2, float *bx3,
        float *by1, float *by2, float *by3,
        float *bz1, float *bz2, float *bz3,
        float *dvx, float *dvy, float *dvz,
        int iNx, int iNy, int iNz){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        
        dvx[idx] = bx1[idx+iNy] - bx1[idx]
                + bx2[idx+1] - bx2[idx]
                + bx3[idx+(iNx*iNy)] - bx3[idx];
        
        dvy[idx] = by1[idx+iNy] - by1[idx]
                + by2[idx+1] - by2[idx]
                + by3[idx+(iNx*iNy)] - by3[idx];
        
        dvz[idx] = bz1[idx+iNy] - bz1[idx]
                + bz2[idx+1] - bz2[idx]
                + bz3[idx+(iNx*iNy)] - bz3[idx];        
    }
}

static __global__ void krnl_8(float *U1x, float *U1y, float *U1z, float *U2x, float *U2y, float *U2z,
        float *q1, float *q2,
        float *G1x, float *G1y, float *G1z, float *G2x, float *G2y, float *G2z,
        float *dv1x, float *dv1y, float *dv1z, float *dv2x, float *dv2y, float *dv2z,
        float *u1x, float *u1y, float *u1z, float *u2x, float *u2y, float *u2z,    
        float cc, float fPenalty, int iNx, int iNy, int iNz, float *d_qq1, float *d_qq2, float *d_qq3){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        

        d_qq1[idx] = U1x[idx] - U2x[idx]
                    + cc*(q2[idx]*G2x[idx] + dv2x[idx] - u2x[idx] - 2*U2x[idx])
                    - cc*(q1[idx]*G1x[idx] + dv1x[idx] - u1x[idx] - 2*U1x[idx]);
        d_qq1[idx] /=  (2*cc);
        d_qq1[idx] = MAX(MIN(d_qq1[idx],fPenalty),-fPenalty);         

        d_qq2[idx] = U1y[idx] - U2y[idx]
                    + cc*(q2[idx]*G2y[idx] + dv2y[idx] - u2y[idx] - 2*U2y[idx])
                    - cc*(q1[idx]*G1y[idx] + dv1y[idx] - u1y[idx] - 2*U1y[idx]);
        d_qq2[idx] /=  (2*cc);
        d_qq2[idx] = MAX(MIN(d_qq2[idx],fPenalty),-fPenalty);  

        d_qq3[idx] = U1z[idx] - U2z[idx]
                    + cc*(q2[idx]*G2z[idx] + dv2z[idx] - u2z[idx] - 2*U2z[idx])
                    - cc*(q1[idx]*G1z[idx] + dv1z[idx] - u1z[idx] - 2*U1z[idx]);
        d_qq3[idx] /=  (2*cc);
        d_qq3[idx] = MAX(MIN(d_qq3[idx],fPenalty),-fPenalty);  
    }
}


static __global__ void krnl_9(float *dv1x, float *dv1y, float *dv1z, float *dv2x, float *dv2y, float *dv2z,
        float *G1x, float *G1y, float *G1z, float *G2x, float *G2y, float *G2z,
        float *q1, float *q2, float *u1x, float *u1y, float *u1z, float *u2x, float *u2y, float *u2z, 
        float *FPS, float  cc, int iNx, int iNy, int iNz, float *d_qq1, float *d_qq2, float *d_qq3){
    
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));
    if (idx >= iNx*iNy*iNz) return;
    
    if( ( (idx%iNy) != (iNy-1) ) &&
            ( (idx/(iNx*iNy)) < (iNz-1) ) &&
            ( ((idx/iNy)%iNx) != (iNx-1))
            ){
        
        /* update ux */
        float fp1t = cc*(dv1x[idx] + q1[idx]*G1x[idx] + d_qq1[idx]);
        float fp2t = cc*(dv2x[idx] + q2[idx]*G2x[idx] - d_qq1[idx]);
        FPS[idx] = fabsf(fp1t) + fabsf(fp2t);
        
        u1x[idx] -= fp1t;
        u2x[idx] -= fp2t;

        /* update uy */
        
        fp1t = cc*(dv1y[idx] + q1[idx]*G1y[idx] + d_qq2[idx]);
        fp2t = cc*(dv2y[idx] + q2[idx]*G2y[idx] - d_qq2[idx]);
        FPS[idx] += fabsf(fp1t) + fabsf(fp2t);
        u1y[idx]  -= fp1t;
        u2y[idx]  -= fp2t;

        /* update uz */
        
        fp1t = cc*(dv1z[idx] + q1[idx]*G1z[idx] + d_qq3[idx]);
        fp2t = cc*(dv2z[idx] + q2[idx]*G2z[idx] - d_qq3[idx]);
        FPS[idx] += fabsf(fp1t) + fabsf(fp2t);
        u1z[idx] -= fp1t;
        u2z[idx] -= fp2t;
    }
}

